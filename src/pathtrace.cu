#include "hip/hip_runtime.h"
#include "pathtrace.h"

#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/remove.h>
#include <thrust/device_vector.h>
#include <thrust/partition.h>

#include "sceneStructs.h"
#include "scene.h"
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"
#include "utilities.h"
#include "intersections.h"
#include "interactions.h"

#define ERRORCHECK 1

#define FILENAME (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)
#define checkCUDAError(msg) checkCUDAErrorFn(msg, FILENAME, __LINE__)
void checkCUDAErrorFn(const char* msg, const char* file, int line)
{
#if ERRORCHECK
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (hipSuccess == err)
    {
        return;
    }

    fprintf(stderr, "CUDA error");
    if (file)
    {
        fprintf(stderr, " (%s:%d)", file, line);
    }
    fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
#ifdef _WIN32
    getchar();
#endif // _WIN32
    exit(EXIT_FAILURE);
#endif // ERRORCHECK
}

__host__ __device__
thrust::default_random_engine makeSeededRandomEngine(int iter, int index, int depth)
{
    int h = utilhash((1 << 31) | (depth << 22) | iter) ^ utilhash(index);
    return thrust::default_random_engine(h);
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* pbo, glm::ivec2 resolution, int iter, glm::vec3* image)
{
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < resolution.x && y < resolution.y)
    {
        int index = x + (y * resolution.x);
        glm::vec3 pix = image[index];

        glm::ivec3 color;
        color.x = glm::clamp((int)(pix.x / iter * 255.0), 0, 255);
        color.y = glm::clamp((int)(pix.y / iter * 255.0), 0, 255);
        color.z = glm::clamp((int)(pix.z / iter * 255.0), 0, 255);

        // Each thread writes one pixel location in the texture (textel)
        pbo[index].w = 0;
        pbo[index].x = color.x;
        pbo[index].y = color.y;
        pbo[index].z = color.z;
    }
}

static Scene* hst_scene = NULL;
static GuiDataContainer* guiData = NULL;
static glm::vec3* dev_image = NULL;
static Geom* dev_geoms = NULL;
static Material* dev_materials = NULL;
static PathSegment* dev_paths = NULL;
static ShadeableIntersection* dev_intersections = NULL;
// TODO: static variables for device memory, any extra info you need, etc
// ...
// Mesh loading variables.
static Vertex* dev_vertices = NULL;

void InitDataContainer(GuiDataContainer* imGuiData)
{
    guiData = imGuiData;
}

void pathtraceInit(Scene* scene)
{
    hst_scene = scene;

    const Camera& cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

    hipMalloc(&dev_image, pixelcount * sizeof(glm::vec3));
    hipMemset(dev_image, 0, pixelcount * sizeof(glm::vec3));

    hipMalloc(&dev_paths, pixelcount * sizeof(PathSegment));

    hipMalloc(&dev_geoms, scene->geoms.size() * sizeof(Geom));
    hipMemcpy(dev_geoms, scene->geoms.data(), scene->geoms.size() * sizeof(Geom), hipMemcpyHostToDevice);

    hipMalloc(&dev_materials, scene->materials.size() * sizeof(Material));
    hipMemcpy(dev_materials, scene->materials.data(), scene->materials.size() * sizeof(Material), hipMemcpyHostToDevice);

    hipMalloc(&dev_intersections, pixelcount * sizeof(ShadeableIntersection));
    hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

    // TODO: initialize any extra device memeory you need
	hipMalloc(&dev_vertices, scene->vertices.size() * sizeof(Vertex));
    hipMemcpy(dev_vertices, scene->vertices.data(), scene->vertices.size() * sizeof(Vertex), hipMemcpyHostToDevice);

    checkCUDAError("pathtraceInit");
}

void pathtraceFree()
{
    hipFree(dev_image);  // no-op if dev_image is null
    hipFree(dev_paths);
    hipFree(dev_geoms);
    hipFree(dev_materials);
    hipFree(dev_intersections);
    // TODO: clean up any extra device memory you created
	hipFree(dev_vertices);

    checkCUDAError("pathtraceFree");
}

__device__ glm::vec3 sampleAperture(float apertureRadius, thrust::default_random_engine& rng) {
    thrust::uniform_real_distribution<float> u01(0, 1);
    float r = apertureRadius * sqrt(u01(rng));
    float theta = 2 * PI * u01(rng);
    // Lens offset.
    return glm::vec3(r * cos(theta), r * sin(theta), 0.0f);
}

/**
* Generate PathSegments with rays from the camera through the screen into the
* scene, which is the first bounce of rays.
*
* Antialiasing - add rays for sub-pixel sampling
* motion blur - jitter rays "in time"
* lens effect - jitter ray origin positions based on a lens
*/
__global__ void generateRayFromCamera(Camera cam, int iter, int traceDepth, PathSegment* pathSegments)
{
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < cam.resolution.x && y < cam.resolution.y) {
        int index = x + (y * cam.resolution.x);
        PathSegment& segment = pathSegments[index];

        //segment.ray.origin = cam.position;
        //segment.color = glm::vec3(1.0f, 1.0f, 1.0f);

        thrust::default_random_engine rng = makeSeededRandomEngine(iter, index, 0);
        thrust::uniform_real_distribution<float> u01(0, 1);

        // Create jitter.
        float jitterX = u01(rng);
        float jitterY = u01(rng);

        // TODO: implement antialiasing by jittering the ray
        //segment.ray.direction = glm::normalize(cam.view
        //    - cam.right * cam.pixelLength.x * ((float)x + jitterX - (float)cam.resolution.x * 0.5f)
        //    - cam.up * cam.pixelLength.y * ((float)y + jitterY - (float)cam.resolution.y * 0.5f)
        //);

        // Redo ray direction calc w/ depth of field.
        glm::vec3 pixelPoint = cam.view
            - cam.right * cam.pixelLength.x * ((float)x + jitterX - (float)cam.resolution.x * 0.5f)
            - cam.up * cam.pixelLength.y * ((float)y + jitterY - (float)cam.resolution.y * 0.5f);
        glm::vec3 rayDir = glm::normalize(pixelPoint);

        // Compute focal point.
        glm::vec3 focalPoint = cam.position + rayDir * cam.focalDist;

        // Depth of field: sample aperture.
        glm::vec3 apertureOffset = sampleAperture(cam.aperture, rng);

        // Updated ray calcs based on depth of field aperture.
        segment.ray.origin = cam.position + apertureOffset;
        segment.color = glm::vec3(1.f);
        segment.ray.direction = glm::normalize(focalPoint - segment.ray.origin);

        segment.pixelIndex = index;
        segment.remainingBounces = traceDepth;
    }
}

// For mesh loading.
// Triangle intersection.
// Moller-Trumbore: returns true if intersect, sets t,u,v.
__host__ __device__ bool intersectTriangle(
    const Ray& r,
    const glm::vec3& v0,
    const glm::vec3& v1,
    const glm::vec3& v2,
    float& tOut, float& uOut, float& vOut)
{
    glm::vec3 edge1 = v1 - v0;
    glm::vec3 edge2 = v2 - v0;
    glm::vec3 pvec = glm::cross(r.direction, edge2);
    float det = glm::dot(edge1, pvec);
    if (fabs(det) < BABY_EPSILON) {
        return false;
    }
    float invDet = 1.0f / det;

    glm::vec3 tvec = r.origin - v0;
    float u = glm::dot(tvec, pvec) * invDet;
    if (u < 0.0f || u > 1.0f) {
        return false;
    }

    glm::vec3 qvec = glm::cross(tvec, edge1);
    float v = glm::dot(r.direction, qvec) * invDet;
    if (v < 0.0f || (u + v) > 1.0f) return false;

    float t = glm::dot(edge2, qvec) * invDet;
    if (t <= BABY_EPSILON) {
        return false;
       }

    tOut = t; uOut = u; vOut = v;
    return true;
}

// TODO:
// computeIntersections handles generating ray intersections ONLY.
// Generating new rays is handled in your shader(s).
// Feel free to modify the code below.
__global__ void computeIntersections(
    int depth,
    int num_paths,
    PathSegment* pathSegments,
    Geom* geoms,
    int geoms_size,
    ShadeableIntersection* intersections,
    Vertex* vertices,
    int numVerts)
{
    int path_index = blockIdx.x * blockDim.x + threadIdx.x;

    if (path_index < num_paths)
    {
        PathSegment pathSegment = pathSegments[path_index];

        float t;
        glm::vec3 intersect_point;
        glm::vec3 normal;
        float t_min = FLT_MAX;
        int hit_geom_index = -1; // -1 for cube, sphere, -2 for triangle, 0 o.w.
        int hit_material_id = -1; 
        bool outside = true;

        glm::vec3 tmp_intersect;
        glm::vec3 tmp_normal;

        // naive parse through global geoms

        for (int i = 0; i < geoms_size; i++)
        {
            Geom& geom = geoms[i];

            if (geom.type == CUBE)
            {
                t = boxIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
            }
            else if (geom.type == SPHERE)
            {
                t = sphereIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
            }
            // TODO: add more intersection tests here... triangle? metaball? CSG?

            // Compute the minimum t from the intersection tests to determine what
            // scene geometry object was hit first.
            if (t > 0.0f && t_min > t)
            {
                t_min = t;
                hit_geom_index = geoms[i].materialid;
                intersect_point = tmp_intersect;
                normal = tmp_normal;
            }
        }

        for (int i = 0; i < numVerts; i += 3) {
            Vertex& v0 = vertices[i + 0];
            Vertex& v1 = vertices[i + 1];
            Vertex& v2 = vertices[i + 2];

            // Call intersectTriangle for mesh triangle intersection.
            float t, u, v;
            if (intersectTriangle(pathSegment.ray, v0.position, v1.position, v2.position, t, u, v)) {
                if (t > 0.0f && t < t_min) {
                    t_min = t;
                    hit_geom_index = -2; 
                    intersect_point = pathSegment.ray.origin + t * pathSegment.ray.direction;

                    // Interpolate normals (if available).
                    if (length(v0.normal) < 1e-6f || length(v1.normal) < 1e-6f || length(v2.normal) < 1e-6f) {
                        normal = normalize(cross(v1.position - v0.position, v2.position - v0.position));
                    }
                    else {
                        normal = normalize((1 - u - v) * v0.normal + u * v1.normal + v * v2.normal);
                    }

                    // Store material directly from vertex (assuming per-triangle consistent).
                    hit_material_id = v0.materialID;
                }
            }
        }

        if (hit_geom_index == -1)
        {
            intersections[path_index].t = -1.0f;
        }
        else
        {
            if (glm::dot(pathSegment.ray.direction, normal) > 0.0f) {
                normal = -normal; // flip to make it face the ray origin.
            }
            // The ray hits something
            intersections[path_index].t = t_min;
            intersections[path_index].materialId = (hit_geom_index == -2) ? hit_material_id : hit_geom_index;
            intersections[path_index].surfaceNormal = normal;
        }
    }
}

// LOOK: "fake" shader demonstrating what you might do with the info in
// a ShadeableIntersection, as well as how to use thrust's random number
// generator. Observe that since the thrust random number generator basically
// adds "noise" to the iteration, the image should start off noisy and get
// cleaner as more iterations are computed.
//
// Note that this shader does NOT do a BSDF evaluation!
// Your shaders should handle that - this can allow techniques such as
// bump mapping.
__global__ void shadeFakeMaterial(
    int iter,
    int num_paths,
    ShadeableIntersection* shadeableIntersections,
    PathSegment* pathSegments,
    Material* materials)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_paths)
    {
        ShadeableIntersection intersection = shadeableIntersections[idx];
        if (intersection.t > 0.0f) // if the intersection exists...
        {
          // Set up the RNG
          // LOOK: this is how you use thrust's RNG! Please look at
          // makeSeededRandomEngine as well.
            thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, 0);
            thrust::uniform_real_distribution<float> u01(0, 1);

            Material material = materials[intersection.materialId];
            glm::vec3 materialColor = material.color;

            // If the material indicates that the object was a light, "light" the ray
            if (material.emittance > 0.0f) {
                pathSegments[idx].color *= (materialColor * material.emittance);
            }
            // Otherwise, do some pseudo-lighting computation. This is actually more
            // like what you would expect from shading in a rasterizer like OpenGL.
            // TODO: replace this! you should be able to start with basically a one-liner
            else {
                float lightTerm = glm::dot(intersection.surfaceNormal, glm::vec3(0.0f, 1.0f, 0.0f));
                pathSegments[idx].color *= (materialColor * lightTerm) * 0.3f + ((1.0f - intersection.t * 0.02f) * materialColor) * 0.7f;
                pathSegments[idx].color *= u01(rng); // apply some noise because why not
            }
            // If there was no intersection, color the ray black.
            // Lots of renderers use 4 channel color, RGBA, where A = alpha, often
            // used for opacity, in which case they can indicate "no opacity".
            // This can be useful for post-processing and image compositing.
        }
        else {
            pathSegments[idx].color = glm::vec3(0.0f);
        }
    }
}

__global__ void kernShadeMaterialProper(
    int iter,
    int num_paths,
    ShadeableIntersection* shadeableIntersections,
    PathSegment* pathSegments,
    Material* materials)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_paths)
    {
        // Check if ray terminated.
        if (pathSegments[idx].remainingBounces <= 0) {
            return;
        }

        ShadeableIntersection intersection = shadeableIntersections[idx];


        if (intersection.t > 0.0f) // if the intersection exists...
        {

            Material material = materials[intersection.materialId];
            glm::vec3 materialColor = material.color;

            // If the material indicates that the object was a light, "light" the ray
            if (material.emittance > 0.0f) {
                // Is light.
                pathSegments[idx].color *= (materialColor * material.emittance);
                pathSegments[idx].remainingBounces = 0;
            }
            
            else {
                // Use thrust to scatter ray.
                thrust::default_random_engine rng = makeSeededRandomEngine(iter, pathSegments[idx].pixelIndex, pathSegments[idx].remainingBounces);
                thrust::uniform_real_distribution<float> u01(0, 1);

                // Get the ray
                Ray& ray = pathSegments[idx].ray;

                // Get the intersection point
                glm::vec3 intersect = ray.origin + ray.direction * intersection.t;

                scatterRay(pathSegments[idx], intersect, intersection.surfaceNormal, material, rng);
            }
            // If there was no intersection, color the ray black.
            // Lots of renderers use 4 channel color, RGBA, where A = alpha, often
            // used for opacity, in which case they can indicate "no opacity".
            // This can be useful for post-processing and image compositing.
        }
        else {
            pathSegments[idx].color = glm::vec3(0.0f);
            pathSegments[idx].remainingBounces = 0;
        }
    }
}

// Add the current iteration's output to the overall image
__global__ void finalGather(int nPaths, glm::vec3* image, PathSegment* iterationPaths)
{
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (index < nPaths)
    {
        PathSegment iterationPath = iterationPaths[index];
        image[iterationPath.pixelIndex] += iterationPath.color;
    }
}

/**
 * Wrapper for the __global__ call that sets up the kernel calls and does a ton
 * of memory management
 */
void pathtrace(uchar4* pbo, int frame, int iter)
{
    const int traceDepth = hst_scene->state.traceDepth;
    const Camera& cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

    // 2D block for generating ray from camera
    const dim3 blockSize2d(8, 8);
    const dim3 blocksPerGrid2d(
        (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
        (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

    // 1D block for path tracing
    const int blockSize1d = 128;

    ///////////////////////////////////////////////////////////////////////////

    // Recap:
    // * Initialize array of path rays (using rays that come out of the camera)
    //   * You can pass the Camera object to that kernel.
    //   * Each path ray must carry at minimum a (ray, color) pair,
    //   * where color starts as the multiplicative identity, white = (1, 1, 1).
    //   * This has already been done for you.
    // * For each depth:
    //   * Compute an intersection in the scene for each path ray.
    //     A very naive version of this has been implemented for you, but feel
    //     free to add more primitives and/or a better algorithm.
    //     Currently, intersection distance is recorded as a parametric distance,
    //     t, or a "distance along the ray." t = -1.0 indicates no intersection.
    //     * Color is attenuated (multiplied) by reflections off of any object
    //   * TODO: Stream compact away all of the terminated paths.
    //     You may use either your implementation or `thrust::remove_if` or its
    //     cousins.
    //     * Note that you can't really use a 2D kernel launch any more - switch
    //       to 1D.
    //   * TODO: Shade the rays that intersected something or didn't bottom out.
    //     That is, color the ray by performing a color computation according
    //     to the shader, then generate a new ray to continue the ray path.
    //     We recommend just updating the ray's PathSegment in place.
    //     Note that this step may come before or after stream compaction,
    //     since some shaders you write may also cause a path to terminate.
    // * Finally, add this iteration's results to the image. This has been done
    //   for you.

    // TODO: perform one iteration of path tracing

    generateRayFromCamera<<<blocksPerGrid2d, blockSize2d>>>(cam, iter, traceDepth, dev_paths);
    checkCUDAError("generate camera ray");

    int depth = 0;
    PathSegment* dev_path_end = dev_paths + pixelcount;
    int num_paths = dev_path_end - dev_paths;

    // --- PathSegment Tracing Stage ---
    // Shoot ray into scene, bounce between objects, push shading chunks

    bool iterationComplete = false;
    while (!iterationComplete)
    {
        // clean shading chunks
        hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

        // tracing
        dim3 numblocksPathSegmentTracing = (num_paths + blockSize1d - 1) / blockSize1d;
        computeIntersections<<<numblocksPathSegmentTracing, blockSize1d>>> (
            depth,
            num_paths,
            dev_paths,
            dev_geoms,
            hst_scene->geoms.size(),
            dev_intersections,
            dev_vertices,
			hst_scene->vertices.size()
        );
        checkCUDAError("trace one bounce");
        hipDeviceSynchronize();
        depth++;

        // TODO:
        // --- Shading Stage ---
        // Shade path segments based on intersections and generate new rays by
        // evaluating the BSDF.
        // Start off with just a big kernel that handles all the different
        // materials you have in the scenefile.
        // TODO: compare between directly shading the path segments and shading
        // path segments that have been reshuffled to be contiguous in memory.


#define MATERIAL_SORTING 0
#if MATERIAL_SORTING
        // Sort the paths by material via stream compaction (thrust).
        thrust::device_ptr<ShadeableIntersection> dev_intersections_ptr(dev_intersections);
        thrust::device_ptr<PathSegment> dev_paths_ptr(dev_paths);
        thrust::stable_sort_by_key(dev_intersections_ptr, dev_intersections_ptr + num_paths, dev_paths_ptr, CompareMat());
#endif

        //shadeFakeMaterial << <numblocksPathSegmentTracing, blockSize1d >> > (
        kernShadeMaterialProper <<<numblocksPathSegmentTracing, blockSize1d>>>(
            iter,
            num_paths,
            dev_intersections,
            dev_paths,
            dev_materials
        );
        hipDeviceSynchronize();

#define STREAM_COMPACTION 1
#if STREAM_COMPACTION
        // Call thrust for stream compaction.
        thrust::device_ptr<PathSegment> dev_thrust_paths(dev_paths);
        thrust::device_ptr<PathSegment> dev_new_ends =
            thrust::stable_partition(thrust::device, dev_thrust_paths, dev_thrust_paths + num_paths, PathAlive());

        num_paths = dev_new_ends.get() - dev_paths;
#endif

        // TODO: should be based off stream compaction results.
        if (num_paths == 0) {
            iterationComplete = true;
        }
        if (depth >= traceDepth) {
            iterationComplete = true;
        }

        if (guiData != NULL)
        {
            guiData->TracedDepth = depth;
        }
    }

    // Assemble this iteration and apply it to the image
    dim3 numBlocksPixels = (pixelcount + blockSize1d - 1) / blockSize1d;
    finalGather<<<numBlocksPixels, blockSize1d>>>(pixelcount, dev_image, dev_paths);

    ///////////////////////////////////////////////////////////////////////////

    // Send results to OpenGL buffer for rendering
    sendImageToPBO<<<blocksPerGrid2d, blockSize2d>>>(pbo, cam.resolution, iter, dev_image);

    // Retrieve image from GPU
    hipMemcpy(hst_scene->state.image.data(), dev_image,
        pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);

    checkCUDAError("pathtrace");
}
